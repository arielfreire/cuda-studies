#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void transpose(int* matriz, int* transposed, int size)
{
    printf("Block: %d, Thread: %d, Block Dim: %d\n", threadIdx.x, blockIdx.x, blockDim.x);
    int column = threadIdx.x;
    int row = blockIdx.x;
    transposed[row * size + column] = matriz[column * size + row];
}

int main() {

    hipError_t cudaStatus;
    const int size = 3;
    const int bytes = size * size * sizeof(int);
    
    int matriz[size][size] = { {1,2,3}, {4,5,6}, {7,8,9} };
    int transposed[size][size] = { {0} };

    int* device_matriz = 0;
    int* device_transposed = 0;

    cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc(&device_matriz, bytes);
    cudaStatus = hipMalloc(&device_transposed, bytes);
    
    cudaStatus = hipMemcpy(device_matriz, matriz, bytes, hipMemcpyHostToDevice);

    transpose << <size, size >> > (device_matriz, device_transposed, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("transpose function failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(transposed, device_transposed, bytes, hipMemcpyDeviceToHost);


    printf("Original Matrix:\n");
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf(" %d ", matriz[i][j]);
        }
        printf("\n");
    }

    printf("Transposed Matrix:\n");
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf(" %d ", transposed[i][j]);
        }
        printf("\n");
    }

    hipFree(device_matriz);
    hipFree(device_transposed);
    return 0;
}