﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

int error(int* device_a, int* device_b, int* device_c);

__global__ void add(int *c, const int *a, const int *b)
{
    printf("Block: %d, Thread: %d, Block Dim: %d\n", threadIdx.x, blockIdx.x, blockDim.x);
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{   
    hipError_t cudaStatus;
   
    const int arraySize = 7;
    const int a[arraySize] = { 1, 2, 3, 5, 7, 11, 13 };
    const int b[arraySize] = { 1, 2, 3, 5, 8, 13, 21 };
    int c[arraySize] = { 0 };

    int* device_a = 0;
    int* device_b = 0;
    int* device_c = 0;
    
    // Seleciona o dispositivo
    cudaStatus = hipSetDevice(0);

    //Aloca memoria na GPU
    cudaStatus = hipMalloc((void**)&device_a, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&device_b, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&device_c, arraySize * sizeof(int));

    cudaStatus = hipMemcpy(device_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(device_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice falhou!");
        return error(device_a, device_b, device_c);
    }
   
    add<<<1, arraySize>>>(device_c, device_a, device_b);
   
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("add function failed: %s\n", hipGetErrorString(cudaStatus));
        return error(device_a, device_b, device_c);
    }
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(c, device_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    printf("{ 1, 2, 3, 5, 7, 11, 13 } + { 1, 2, 3, 5, 8, 13, 21 } = {%d, %d, %d, %d, %d, %d, %d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    return 0;
}

int error(int* device_a, int* device_b, int* device_c) {
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 1;
}

